
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#define MAXLINESIZE 100

__device__ int dNUM;
__device__ int dBLOCK_N;
__device__ int dTHREAD_N;
__device__ float dx1;
__device__ float dy1;
__device__ float dx2;
__device__ float dy2;
__device__ float dx3;
__device__ float dy3;

// NUmber of threads = 96, Number of SM = 2, Number of cores per SM = 48 
__global__ 
void calculate_triliteration(float *dda, float *ddb, float *ddc, float *dx, float *dy){
	extern __shared__ float temp[];
	float *tempX = temp;
	float *tempY = (float *)&tempX[dTHREAD_N];
	
	int blockId = blockIdx.x;
	int threadId = threadIdx.x;
	
	int idx = blockId * dTHREAD_N + threadId;
	int totalThreads = dTHREAD_N * dBLOCK_N;
	
	float dx1dx2 = pow(dx1, 2) - pow(dx2, 2);
	float dy1dy2 = pow(dy1, 2) - pow(dy2, 2);
	
	float dx1dx3 = pow(dx1, 2) - pow(dx3, 2);
	float dy1dy3 = pow(dy1, 2) - pow(dy3, 2);
	
	float dy2dy1 = 2 * (dy2 - dy1);
	float dy3dy1 = 2 * (dy3 - dy1);
	
	float dx2dx1 = 2 * (dx2 - dx1);
	float dx3dx1 = 2 * (dx3 - dx1);
	
	float denX = dx2dx1 * dy3dy1 - dx3dx1 * dy2dy1; 	
	
	
	 
	for(int i = idx; i < dNUM; i += totalThreads){
		if(denX == 0){
			if(i % 4 == 0){
				dx[i/4] = 0;
				dy[i/4] = 0;		
			}
			return;
		}
		float ddaddb = pow(dda[i], 2) - pow(ddb[i], 2);
		float ddaddc = pow(dda[i], 2) - pow(ddc[i], 2);
		
		float numX = (ddaddb - dx1dx2 - dy1dy2) * dy3dy1 - (ddaddc - dx1dx3 - dy1dy3) * dy2dy1;
		float numY = (ddaddc - dx1dx3 - dy1dy3) * dx2dx1 - (ddaddb - dx1dx2 - dy1dy2) * dx3dx1;
		tempX[threadId] = numX / denX;
		tempY[threadId] = numY / denX;
/*		printf("The output is => %f %f\n", tempX[threadId], tempY[threadId]);*/
		__syncthreads();
		if(i % 4 == 0){
			dx[i/4] = (tempX[threadId] + tempX[threadId + 1] + tempX[threadId + 2] + tempX[threadId + 3] ) / 4;
			dy[i/4] = (tempY[threadId] + tempY[threadId + 1] + tempY[threadId + 2] + tempY[threadId + 3] ) / 4;				
		}
		__syncthreads();
		
	}
}


int main(int args, char ** argv){
	clock_t begin = clock();
	if(args != 6){
		printf("Invalid Arguments\nUsage: ./triliteration <NUM> <BLOCK_N> <THREAD_N> <INPUT_FILE> <CHECK_FILE>\n");	
		return -1;
	}
	int NUM = pow(2, atoi(argv[1]));
	int BLOCK_N = atoi(argv[2]);
	int THREAD_N = atoi(argv[3]);
	char *INPUT_FILE = argv[4];
	char *CHECK_FILE = argv[5];
	printf("\n\tInput Size : %d\n", NUM);
	printf("\tBlock_N  : %d\n", BLOCK_N);
	printf("\tTHREAD_N : %d\n", THREAD_N);
	
	FILE * input_fd, * check_fd;
	input_fd = fopen(INPUT_FILE, "r");
	char line[MAXLINESIZE];
	int line_count = 0;
	float da[NUM],db[NUM],dc[NUM], x[NUM / 4], y[NUM / 4];
	float x1 = 0, y1 = 0, x2 = 0, y2 = 0, x3 = 0, y3 = 0;
	
	if(fgets(line, MAXLINESIZE, input_fd) != NULL){
		sscanf(line, "%f %f %f %f %f %f\n", &x1, &y1, &x2, &y2, &x3, &y3);	
	}
	/*x1 = 0.0;
	y1 = 0.0;
	x2 = 20000.0;
	y2 = 20000.0;
	x3 = 40000.0;
	y3 = 0.0;*/
	printf("\nThe Guard points:\n\t%f, %f \n\t%f, %f\n\t%f, %f\n", x1, y1, x2, y2, x3, y3);
	
	while ( fgets(line, MAXLINESIZE, input_fd) != NULL && line_count < NUM) {
		da[line_count] = 0.0;
		db[line_count] = 0.0;
		dc[line_count] = 0.0;
		/*printf("The line is %s\n",line );*/
		sscanf(line, "%f %f %f\n", &da[line_count], &db[line_count], &dc[line_count]);
		/*printf("the values are => %f %f %f\n", da[line_count], db[line_count], dc[line_count]);*/
		//da[line_count] = 3.0; db[line_count] = 3.0; dc[line_count] = 3.0;
		line_count ++;
	}
		
	float * dda, * ddb, * ddc;
	float * dx, * dy;
	int _floatSize = NUM * sizeof(float);
	hipMalloc( (void**)&dda, _floatSize);
	hipMalloc( (void**)&ddb, _floatSize);
	hipMalloc( (void**)&ddc, _floatSize);
	hipMalloc( (void**)&dx, _floatSize / 4);
	hipMalloc( (void**)&dy, _floatSize / 4);
	
	hipMemcpyToSymbol(HIP_SYMBOL(dNUM), &NUM, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(dBLOCK_N), &BLOCK_N, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(dTHREAD_N), &THREAD_N, sizeof(int));
	
	hipMemcpy( dda, da, _floatSize, hipMemcpyHostToDevice ); 
	hipMemcpy( ddb, db, _floatSize, hipMemcpyHostToDevice );	
	hipMemcpy( ddc, dc, _floatSize, hipMemcpyHostToDevice );
	
	hipMemcpyToSymbol( HIP_SYMBOL(dx1), &x1, sizeof(float));
	hipMemcpyToSymbol( HIP_SYMBOL(dy1), &y1, sizeof(float));
	hipMemcpyToSymbol( HIP_SYMBOL(dx2), &x2, sizeof(float));
	hipMemcpyToSymbol( HIP_SYMBOL(dy2), &y2, sizeof(float));
	hipMemcpyToSymbol( HIP_SYMBOL(dx3), &x3, sizeof(float));
	hipMemcpyToSymbol( HIP_SYMBOL(dy3), &y3, sizeof(float));
	
	calculate_triliteration<<<BLOCK_N, THREAD_N, 2 * THREAD_N * sizeof(float)>>>(dda, ddb, ddc, dx, dy);
	hipMemcpy( x, dx, _floatSize / 4, hipMemcpyDeviceToHost);
	hipMemcpy( y, dy, _floatSize / 4, hipMemcpyDeviceToHost);
	printf("Legend : \n\t. => difference less than 0.1,\n\tX => difference greater than 0.1 \n");
	check_fd = fopen(CHECK_FILE, "r");
	float avgX = 0.0, avgY= 0.0;
	//printf("Original X and Y\t Calculated X and Y\n");
	for(int i = 0; i < NUM / 4; i ++){
		fgets(line, MAXLINESIZE, check_fd);
		sscanf(line, "%f %f", &avgX, &avgY);
		if(abs(x[i] - avgX) <= 0.1 && abs(y[i] - avgY) <= 0.1){
			printf(".");	
		}else{
			printf("X");		
		}
		//printf("%f, %f\t%f, %f\n", x[i], y[i], avgX, avgY);
	
	}
	hipFree(dda);
	hipFree(ddb);
	hipFree(ddc);
	hipFree(dx);
	hipFree(dy);
	
	
	fclose(input_fd);
	fclose(check_fd);
	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	printf("\nExecution time is %f\n", time_spent);
	return 1;
}