
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#define MAXLINESIZE 35

__device__ int dNUM;
__device__ int dBLOCK_N;
__constant__ int dTHREAD_N;

// NUmber of threads = 96, Number of SM = 2, Number of cores per SM = 48 
__global__ 
void calculate_triliteration(float *dda, float *ddb, float *ddc, float *dx0, float *dy0){
	extern __shared__ float temp[];
	int blockId = blockIdx.x;
	int threadId = threadIdx.x;
	int idx = blockId * dTHREAD_N + threadId;
	int totalThreads = dTHREAD_N * dBLOCK_N;
	for(int i = idx; i < dNUM; i += totalThreads){
		//printf("%d\n", i);
		temp[threadId] = dda[i] + ddb[i] + ddc[i];
		__syncthreads();
		if(i % 4 == 0){
			dx0[i/4] = temp[threadId] + temp[threadId + 1] + temp[threadId + 2] + temp[threadId + 3] ;				
		}		
		__syncthreads();
		//dy0[idx] = idx;
	}
}


int main(int args, char ** argv){
	if(args != 5){
		printf("Invalid input.....\n");	
		return -1;
	}
	int NUM = pow(2, atoi(argv[1]));
	int BLOCK_N = atoi(argv[2]);
	int THREAD_N = atoi(argv[3]);
	char *INPUT_FILE = argv[4];
	printf("\n\tInput Size : %d\n", NUM);
	printf("\tBlock_N  : %d\n", BLOCK_N);
	printf("\tTHREAD_N : %d\n", THREAD_N);
	
	FILE * input_fd;
	input_fd = fopen(INPUT_FILE, "r");
	char line[MAXLINESIZE];
	int line_count = 0;
	float da[NUM],db[NUM],dc[NUM], x0[NUM / 4], y0[NUM/ 4];
	while ( fgets(line, MAXLINESIZE, input_fd) != NULL && line_count < NUM) {
		da[line_count] = 0.0;
		db[line_count] = 0.0;
		dc[line_count] = 0.0;
		sscanf(line, "%f %f %f\n", &da[line_count], &db[line_count], &dc[line_count]);
		da[line_count] = 1.0; db[line_count] = 1.0; dc[line_count] = 1.0;
		//printf("da = %f, db = %f, dc = %f\n", da[line_count], db[line_count], dc[line_count]);
		line_count ++;
	}
		
	float * dda, * ddb, * ddc;
	float * dx0, * dy0;
	int _floatSize = NUM * sizeof(float);
	hipMalloc( (void**)&dda, _floatSize);
	hipMalloc( (void**)&ddb, _floatSize);
	hipMalloc( (void**)&ddc, _floatSize);
	hipMalloc( (void**)&dx0, _floatSize / 4);
	hipMalloc( (void**)&dy0, _floatSize / 4);
	hipMemcpyToSymbol(HIP_SYMBOL(dNUM), &NUM, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(dBLOCK_N), &BLOCK_N, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(dTHREAD_N), &THREAD_N, sizeof(int));
	
	hipMemcpy( dda, da, _floatSize, hipMemcpyHostToDevice ); 
	hipMemcpy( ddb, db, _floatSize, hipMemcpyHostToDevice );	
	hipMemcpy( ddc, dc, _floatSize, hipMemcpyHostToDevice );
	calculate_triliteration<<<BLOCK_N, THREAD_N, THREAD_N * sizeof(float)>>>(dda, ddb, ddc, dx0, dy0);
	hipMemcpy( x0, dx0, _floatSize / 4, hipMemcpyDeviceToHost);
	//cudaMemcpy( y0, dy0, _floatSize, cudaMemcpyDeviceToHost);
	for(int i = 0; i < NUM / 4; i ++){
		printf("%f\n", x0[i]);	
	}
	hipFree(dda);
	hipFree(ddb);
	hipFree(ddc);
	hipFree(dx0);
	hipFree(dy0);
	
	
	fclose(input_fd);
	
	return 1;
}